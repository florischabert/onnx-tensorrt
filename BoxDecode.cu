#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2018, NVIDIA CORPORATION. All rights reserved.
 *
 * Permission is hereby granted, free of charge, to any person obtaining a
 * copy of this software and associated documentation files (the "Software"),
 * to deal in the Software without restriction, including without limitation
 * the rights to use, copy, modify, merge, publish, distribute, sublicense,
 * and/or sell copies of the Software, and to permit persons to whom the
 * Software is furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in
 * all copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT.  IN NO EVENT SHALL
 * THE AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
 * FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER
 * DEALINGS IN THE SOFTWARE.
 */

#include "BoxDecode.hpp"

#include <thrust/gather.h>
#include <thrust/tabulate.h>
#include <cassert>

nvinfer1::Dims BoxDecodePlugin::getOutputDimensions(int index,
                                                    const nvinfer1::Dims *inputDims,
                                                    int nbInputs) {
  assert(nbInputs >= 3);
  assert(index < this->getNbOutputs());
  switch( index ) {
    case 1: // boxes
      return {2, {_detections_per_im, 4}};
    case 3: // batch splits
      return {1, {1}};
    default:// scores, classes
      return {2, {_detections_per_im, 1}};
  }
}

int BoxDecodePlugin::initialize() {
  return 0;
}

int BoxDecodePlugin::enqueue(int batchSize,
                             const void *const *inputs, void **outputs,
                             void *workspace, hipStream_t stream) {
  auto nbInputs = _input_dims.size();
  auto im_info_ptr = static_cast<const float *>(inputs[0]);
  auto scores_ptr = static_cast<float *>(outputs[0]);
  auto classes_ptr = static_cast<float *>(outputs[1]);
  auto boxes_ptr = static_cast<float4 *>(outputs[2]);

  for( int batch = 0; batch < batchSize; batch++ ) {
    thrust::device_vector<float> all_scores(0);
    thrust::device_vector<int> all_classes(0);
    thrust::device_vector<float4> all_boxes(0);

    for( size_t i = 1; i < nbInputs; i += 2 ) {
      auto const& scores_dims = this->getInputDims(i);
      auto scores_ptr = static_cast<const float *>(inputs[i]);
      auto const& boxes_dims = this->getInputDims(i+1);
      auto boxes_ptr = static_cast<const float4 *>(inputs[i+1]);    
    
      int height = scores_dims.d[1];
      int width = scores_dims.d[2];
      int num_anchors = boxes_dims.d[0] / 4; 
      int num_classes = boxes_dims.d[0] / num_anchors;
      int scores_size = batchSize * num_anchors * num_classes * height * width;
    
      // // Filter scores above threshold 
      thrust::device_vector<int> indices(scores_size);
      auto last_idx = thrust::copy_if(
        thrust::make_counting_iterator<int>(0),
        thrust::make_counting_iterator<int>(scores_size),
        thrust::device_pointer_cast(scores_ptr),
        indices.begin(),
        thrust::placeholders::_1 > _score_thresh);
      indices.resize(thrust::distance(indices.begin(), last_idx));
      
      // Gather filtered scores
      thrust::device_vector<float> scores(indices.size());
      thrust::gather(indices.begin(), indices.end(),
        thrust::device_pointer_cast(scores_ptr), scores.begin());

      // Sort scores and corresponding indices
      thrust::sort_by_key(scores.begin(), scores.end(), indices.begin(), 
        thrust::greater<float>());

      auto pre_nms_top_n = std::min(static_cast<int>(indices.size()), _pre_nms_top_n);
      scores.resize(pre_nms_top_n);
      indices.resize(pre_nms_top_n);

      // Gather boxes
      thrust::device_vector<float4> boxes(pre_nms_top_n);
      thrust::gather(indices.begin(), indices.end(),
        thrust::device_pointer_cast(boxes_ptr), boxes.begin());

      // Get classes
      thrust::device_vector<float> classes(indices.size());
      thrust::transform(indices.begin(), indices.end(), classes.begin(),
        (thrust::placeholders::_1 / height / width) % num_classes);

      if( !_anchors.empty() ) {
        // Add anchors offsets to deltas
        auto anchors_ptr = _anchors.data();
        for( size_t c = 0; c < i/2; c++ ) anchors_ptr += _anchors_counts[i/2];
        thrust::device_vector<float> anchors(_anchors_counts[i/2]);
        thrust::copy_n(anchors_ptr, _anchors_counts[i/2], anchors.begin());
        auto anchors_ptr_d = thrust::raw_pointer_cast(anchors.data());
        
        thrust::transform(
          boxes.begin(), boxes.end(), indices.begin(), boxes.begin(),
          [=] __device__ (float4 b, int i) {
            float im_scale = im_info_ptr[0] / height;
            float x = (i % width) * im_scale;
            float y = ((i / width)  % height) * im_scale;
            int a = (i / num_classes / height / width) % num_anchors;
            float *d = anchors_ptr_d + 4*a;
            return float4{x+d[0]+b.x, y+d[1]+b.y, x+d[2]+b.z, y+d[3]+b.w};
          });
      }

      // Expand detections list
      auto size = all_scores.size();
      all_scores.resize(size + scores.size());
      thrust::copy_n(all_scores.begin() , scores.size(), scores.begin());
      thrust::copy_n(all_classes.begin() + size, classes.size(), classes.begin());
      thrust::copy_n(all_boxes.begin() + size, boxes.size(), boxes.begin());
    }

    // Per class non maximum suppression


    all_scores.resize(_detections_per_im);
    all_classes.resize(_detections_per_im);
    all_boxes.resize(_detections_per_im);

    int offset = _detections_per_im * batch;
    thrust::copy(all_scores.begin(), all_scores.end(), 
      thrust::device_pointer_cast(scores_ptr + offset));
    thrust::copy(all_classes.begin(), all_classes.end(), 
      thrust::device_pointer_cast(classes_ptr + offset));
    thrust::copy(all_boxes.begin(), all_boxes.end(), 
      thrust::device_pointer_cast(boxes_ptr + offset));
  }

  return 0;
}
